#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <mma.h>
#include <hip/hip_runtime.h>
#include "gpu-new-forward.h"

using namespace nvcuda;

#define TILE_SIZE 16

__global__ void matmul_conv_fused(const float *mask, const float *input, float *output,
                                       int Batch, int Map_out, int Channel, int Height, int Width, int K){

     /*
    TODO: Modify this function to implement the fused unroll-matmul-permute kernel.
    
    Function parameter definitions:
    mask - convolution kernel
    input - input
    output - output
    Batch - batch_size (number of images in x)
    Map_out - number of output feature maps
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */
    
    int outH = Height-K+1;
    int outW = Width-K+1;
    int Bcols = outH*outW;
    int Acols = Channel*K*K;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bz = blockIdx.z;
    int warpM = (threadIdx.x/32)%2;
    int warpN = (threadIdx.x/32)/2;
    int laneId = threadIdx.x%32;
    constexpr int WMMA_M = 16;
    constexpr int WMMA_N = 16;
    constexpr int WMMA_K = 8;
    int row = by*WMMA_M;
    int col = bx*WMMA_N;
    __shared__ float s_a[WMMA_M*WMMA_K];
    __shared__ float s_b[WMMA_K*WMMA_N];
    __shared__ float s_c[WMMA_M*WMMA_N];
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, wmma::precision::tf32, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, wmma::precision::tf32, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
    wmma::fill_fragment(c_frag, 0.0f);
    int tid = threadIdx.x;

    
    for(int tileK=0; tileK<(Acols+WMMA_K-1)/WMMA_K; ++tileK){
        
        for(int idx=tid; idx<WMMA_M*WMMA_K; idx+=blockDim.x){
            int i = idx/WMMA_K;
            int j = idx%WMMA_K;
            int mask_i = row+i;
            int mask_j = tileK*WMMA_K+j;
            if(mask_i<Map_out && mask_j<Acols){
                s_a[i*WMMA_K+j] = mask[mask_i*Acols+mask_j];
            }
            else{
                s_a[i*WMMA_K+j] = 0.0f;
            }
        }
        for(int idx=tid; idx<WMMA_K*WMMA_N; idx+=blockDim.x){
            int i = idx/WMMA_N;
            int j = idx%WMMA_N;

            int kOffset = tileK*WMMA_K+i;
            if(kOffset<Acols){
                int c = kOffset/(K*K);
                int kIdx = kOffset%(K*K);
                int kh = kIdx/K;
                int kw = kIdx%K;

                int outIdx = col+j;
                int oh = outIdx/outW;
                int ow = outIdx%outW;

                int h = oh+kh;
                int w = ow+kw;

                if(col+j<Bcols && c<Channel && h<Height && w<Width){
                    int input_idx = ((bz*Channel+c)*Height+h)*Width+w;
                    s_b[i*WMMA_N+j] = input[input_idx];
                }else{
                    s_b[i*WMMA_N+j] = 0.0f;
                }
            }else{
                s_b[i*WMMA_N+j] = 0.0f;
            }
        }

        __syncthreads();

        // Perform matrix multiplication using tensor cores
        if(tid<32){
            wmma::load_matrix_sync(a_frag, s_a, WMMA_K);
            wmma::load_matrix_sync(b_frag, s_b, WMMA_N);
            wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
        }

        __syncthreads();
    }

    
    if(tid<32){
        wmma::store_matrix_sync(s_c, c_frag, WMMA_N, wmma::mem_row_major);
    }

    __syncthreads();

   
    for(int idx=tid; idx<WMMA_M*WMMA_N; idx+=blockDim.x){
        int i = idx/WMMA_N;
        int j = idx%WMMA_N;

        int globalRow = row+i;
        int globalCol = col+j;

        if(globalRow<Map_out && globalCol<Bcols){
            output[(bz*Map_out+globalRow)*Bcols+globalCol] = s_c[i*WMMA_N+j];
        }
    }
}


__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask,
                                                    float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr,
                                                    const int Batch, const int Map_out, const int Channel,
                                                    const int Height, const int Width, const int K)
{
    int oh = Height - K + 1;
    int ow = Width - K + 1;
    size_t outS = Batch * Map_out * oh * ow * sizeof(float);
    size_t inS = Batch * Channel * Height * Width * sizeof(float);
    size_t maskS = Map_out * Channel * K * K * sizeof(float);

    hipMalloc((void**) device_output_ptr, outS);
    hipMalloc((void**) device_input_ptr, inS);
    hipMalloc((void**) device_mask_ptr, maskS);

    hipMemcpy(*device_input_ptr, host_input, inS, hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, maskS, hipMemcpyHostToDevice);
}

__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask,
                                             const int Batch, const int Map_out, const int Channel,
                                             const int Height, const int Width, const int K)
{
    // Calculate output dimensions
    int oh = Height - K + 1;
    int ow = Width - K + 1;
    
    // Configure kernel execution parameters for tensor cores
    // Use 128 threads per block (4 warps) for better occupancy
    dim3 blockDim(128, 1, 1);
    
    // Calculate grid dimensions to cover the entire output
    dim3 gridDim((ow * oh + TILE_SIZE - 1) / TILE_SIZE, 
                 (Map_out + TILE_SIZE - 1) / TILE_SIZE, 
                 Batch);

    matmul_conv_fused<<<gridDim, blockDim>>>(device_mask, device_input, device_output,
                                                  Batch, Map_out, Channel, Height, Width, K);
}

__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output,
                                                    float *device_input, float *device_mask,
                                                    const int Batch, const int Map_out, const int Channel,
                                                    const int Height, const int Width, const int K)
{
    int oh = Height - K + 1;
    int ow = Width - K + 1;
    size_t outS = Batch * Map_out * oh * ow * sizeof(float);

    hipMemcpy(host_output, device_output, outS, hipMemcpyDeviceToHost);

    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask);
}

__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}