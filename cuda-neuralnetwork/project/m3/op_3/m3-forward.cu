#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_SIZE 16
#define COARSEN   4

__global__ void matmul_conv_fused(const float *mask, const float *input, float *output,
                                  int Batch, int Map_out, int Channel, int Height, int Width, int K)
{

    /*
    TODO: Modify this function to implement the fused unroll-matmul-permute kernel.
    
    Function parameter definitions:
    mask - convolution kernel
    input - input
    output - output
    Batch - batch_size (number of images in x)
    Map_out - number of output feature maps
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */


    int outH  = Height - K + 1;
    int outW  = Width  - K + 1;
    int Bcols = outH * outW;
    int Acols = Channel * K * K;

    int imgIdx  = blockIdx.z;
    int rowIdx  = blockIdx.y * TILE_SIZE + threadIdx.y;
    int baseCol = blockIdx.x * (TILE_SIZE * COARSEN) + threadIdx.x;

    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];

    float total[COARSEN] = {0.0f};

    int numChunks = (Acols + TILE_SIZE - 1) / TILE_SIZE;

    for (int chunk = 0; chunk < numChunks; ++chunk) {
        int maskCol = chunk * TILE_SIZE + threadIdx.x;
        if(rowIdx < Map_out && maskCol < Acols){
            tileA[threadIdx.y][threadIdx.x] = mask[rowIdx * Acols + maskCol];
        } 
        else{
            tileA[threadIdx.y][threadIdx.x] = 0.0f;
        };

        __syncthreads();

        int inRow = chunk * TILE_SIZE + threadIdx.y;
        int ch    = inRow / (K * K);
        int rem   = inRow % (K * K);
        int p     = rem / K;
        int q     = rem % K;

        for (int c = 0; c < COARSEN; ++c) {
            int colIdx = baseCol + c * TILE_SIZE;
            float v = 0.0f;
            if (inRow < Acols && colIdx < Bcols) {
                int outY = colIdx / outW;
                int outX = colIdx % outW;
                int inY  = outY + p;
                int inX  = outX + q;
                if (inY < Height && inX < Width) {
                    size_t idx = ((size_t)imgIdx * Channel + ch) * Height * Width
                               + inY * Width + inX;
                    v = input[idx];
                }
            }
            tileB[threadIdx.y][threadIdx.x] = v;
            __syncthreads();

            
            for (int k = 0; k < TILE_SIZE; ++k) {
                total[c] += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];
            }
            __syncthreads();
        }
    }

    if (rowIdx < Map_out) {
        for (int c = 0; c < COARSEN; ++c) {
            int colIdx = baseCol + c * TILE_SIZE;
            if (colIdx < Bcols) {
                int outIdx = ((imgIdx * Map_out + rowIdx) * Bcols) + colIdx;
                output[outIdx] = total[c];
            }
        }
    }
}

__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{

    // TODO: Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }


    int oh = Height - K + 1;
    int ow = Width  - K + 1;
    size_t outS  = (size_t)Batch * Map_out * oh * ow * sizeof(float);
    size_t inS   = (size_t)Batch * Channel * Height * Width * sizeof(float);
    size_t maskS = (size_t)Map_out * Channel * K * K * sizeof(float);

    hipMalloc(device_output_ptr, outS);
    hipMalloc(device_input_ptr,  inS);
    hipMalloc(device_mask_ptr,   maskS);

    hipMemcpy(*device_input_ptr, host_input, inS,    hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr,  host_mask,  maskS,  hipMemcpyHostToDevice);
}

__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{

    // TODO: Set the kernel dimensions and call the fused kernel

    int oh = Height - K + 1;
    int ow = Width  - K + 1;
    int Bcols = oh * ow;

    dim3 blockDim(TILE_SIZE, TILE_SIZE, 1);
    dim3 gridDim((Bcols + TILE_SIZE * COARSEN - 1) / (TILE_SIZE * COARSEN),
                 (Map_out + TILE_SIZE - 1) / TILE_SIZE,
                  Batch);

    matmul_conv_fused<<<gridDim, blockDim>>>(
        device_mask, device_input, device_output,
        Batch, Map_out, Channel, Height, Width, K);
}

__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{

    // TODO: Copy the output back to host

    int oh = Height - K + 1;
    int ow = Width  - K + 1;
    size_t outS = (size_t)Batch * Map_out * oh * ow * sizeof(float);

    hipMemcpy(host_output, device_output, outS, hipMemcpyDeviceToHost);
    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask);
}

__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}